#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <stdio.h>
#include <stdlib.h>

#include "include/gradient.cuh"
#include "include/fluid_sim.cuh"
#include "include/fluid_utils.cuh"

static float rand_norm_scalar()
{
    float retval = (float) rand() / (float) RAND_MAX + 1.0;
    retval = retval * ((rand() % 2 > 0) ? 1.0 : -1.0);
    return retval;
}

static __host__ void initialize_p_field(float *data, const MatrixDim dim)
{
    const size_t radius = 256;
    const size_t x_lower = dim.x/2 - radius;
    const size_t x_upper = dim.x/2 + radius;
    const size_t y_lower = dim.y/2 - radius - 100;
    const size_t y_upper = dim.y/2 + radius + 100;

    for (size_t y = y_lower; y < y_upper; y++)
    {
        for (size_t x = x_lower; x < x_upper; x++)
        {
            data[matrix_index(x,y,dim,0)] = 2.0 * rand_norm_scalar();
            data[matrix_index(x,y,dim,1)] = 2.0;// * rand_norm_scalar();
        }
    }
}

static __host__ void initialize_bgr_field(unsigned int *data, const size_t nElements)
{
    // Set the alpha field to max for all pixels
    // Note: little endian representation (A,R,G,B)
    for (size_t idx = 0; idx < nElements; idx++)
        data[idx] = 0xFF000000; 
}

int main()
{
    const MatrixDim DIMENSIONS = {768, 768, 2};
    const size_t N_ELEMENTS = DIMENSIONS.x * DIMENSIONS.y;
    const size_t FIELD_SIZE = sizeof(float) * N_ELEMENTS * DIMENSIONS.vl;
    const size_t BGR_SIZE = sizeof(unsigned int) * N_ELEMENTS;
    const size_t RDX = 512;

    // Simulation timestep
    const float TIMESTEP = 0.01;

    // Rendering frame rate (milliseconds)
    const int FRAMERATE = 1;

    // Setup CUDA Grids and Blocks
    const dim3 DIM_BLOCK(32,32); // This is the maximum as per CUDA 2.x
    const dim3 DIM_GRID(
        (DIMENSIONS.x + DIM_BLOCK.x - 1) / DIM_BLOCK.x,
        (DIMENSIONS.y + DIM_BLOCK.y - 1) / DIM_BLOCK.y);

    // Setup host pressure field
    float *h_pfield;
    hipHostMalloc(&h_pfield, FIELD_SIZE);
    initialize_p_field(h_pfield, DIMENSIONS);

    // Setup device pressure field
    float *d_pfield;
    float *d_pfield_temp;
    hipMalloc(&d_pfield, FIELD_SIZE);
    hipMalloc(&d_pfield_temp, FIELD_SIZE);

    // Setup host image matrix
    unsigned int *h_bgr;
    hipHostMalloc(&h_bgr, BGR_SIZE);
    initialize_bgr_field(h_bgr, N_ELEMENTS);

    // Setup device image matrix
    unsigned int *d_bgr;
    hipMalloc(&d_bgr, BGR_SIZE);
    hipMemcpy(d_bgr, h_bgr, BGR_SIZE, hipMemcpyHostToDevice);

    cv::Mat image;
    size_t temp = 0;
    while (true)
    {
        hipMemcpy(d_pfield, h_pfield, FIELD_SIZE, hipMemcpyHostToDevice);
        hipMemcpy(d_pfield_temp, d_pfield, FIELD_SIZE, hipMemcpyDeviceToDevice);
        kernel_advect<<<DIM_GRID, DIM_BLOCK>>>(DIMENSIONS,d_pfield,d_pfield_temp,RDX,TIMESTEP,0.8);
        hipMemcpy(d_pfield, d_pfield_temp, FIELD_SIZE, hipMemcpyDeviceToDevice);
        kernel_gradient<<<DIM_GRID, DIM_BLOCK>>>(d_pfield, d_bgr, DIMENSIONS);
        hipMemcpy(h_pfield, d_pfield, FIELD_SIZE, hipMemcpyDeviceToHost);
        hipMemcpy(h_bgr, d_bgr, BGR_SIZE, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();

        image = cv::Mat(DIMENSIONS.y, DIMENSIONS.x, CV_8UC4, (unsigned *)h_bgr);
#if 0
        for (size_t idx = 0; idx < N_ELEMENTS; idx++)
        {
            printf("%x\n",h_bgr[idx]);
        }
#endif
#if 0
        printf("iteration: %d\n",temp);
        for (size_t idx = 0; idx < N_ELEMENTS; idx++)
        {
            printf("(%f %f) ", h_pfield[idx * 2], h_pfield[idx * 2+1]);
            if (idx % WIDTH == 0)
                printf("\n");
        }
        for (size_t idx = 0; idx < N_ELEMENTS; idx++)
        {
            printf("0x%x ", h_bgr[idx]);
            if (idx % WIDTH == 0)
                printf("\n");
        }
#endif
        // printf("0x%X\n",normalized2bgr(1.0));
        //printf("0x%X\n", data[1]);
        //printf("%f\n",rand_norm_scalar());

        cv::namedWindow("Display Image", cv::WINDOW_AUTOSIZE);
        cv::imshow("Display Image", image);
        cv::waitKey(FRAMERATE);

        if (temp > 100)
        {
            temp = 0;
            initialize_p_field(h_pfield, DIMENSIONS);
        }
        else
        {
            temp++;
        }
    }

    hipFree(h_pfield);
    hipFree(d_pfield);
    hipFree(h_bgr);
    return 0;
}