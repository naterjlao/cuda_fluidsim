#include "hip/hip_runtime.h"
#include <stdio.h>
#include "include/fluid_sim.cuh"
#include "include/gradient.cuh"

__global__ void thread_idx_2D(size_t *buffer, const size_t nCols)
{
    // Example of row, col access into 2D arrays
    const size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    const size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    buffer[row * nCols + col] = col;
}

void test_2D_dim()
{
    const size_t nRows = 512;
    const size_t nCols = 512;
    size_t buffer[nRows][nCols];
    size_t *d_buffer;

    // Method of calculating multi-dimensional kernel calls
    dim3 dimBlock(32, 32); // This is the maximum as per CUDA 2.x
    dim3 dimGrid( // Method of calculating the number of blocks to use
        (nCols + dimBlock.x - 1) / dimBlock.x,
        (nRows + dimBlock.y - 1) / dimBlock.y);

    memset(buffer, 0, sizeof(buffer));
    hipMalloc(&d_buffer,sizeof(size_t) * nRows *nCols);
    hipMemcpy(d_buffer, buffer, sizeof(buffer), hipMemcpyHostToDevice);
    thread_idx_2D<<<dimGrid,dimBlock>>>(d_buffer, nCols);
    hipMemcpy(buffer, d_buffer, sizeof(buffer), hipMemcpyDeviceToHost);
    for (size_t idx = 0; idx < nRows; idx++)
    {
        for (size_t jdx = 0; jdx < nCols; jdx++)
            printf("%d ", buffer[idx][jdx]);
        printf("\n");
    }
    hipFree(d_buffer);
}

__global__ void thread_idx_ND(size_t *buffer,
    const size_t nRows,
    const size_t nCols,
    const size_t nDims)
{
    // Example of row, col access into 2D arrays
    const size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    const size_t z = blockIdx.z * blockDim.z + threadIdx.z;

    // Conditional guard since the grid may operate
    // out-of-bounds
    if (y < nRows && x < nCols)
        buffer[(y * nCols + x ) * nDims + z] = z;
}

void test_ND_dim()
{
    const size_t nRows = 20;
    const size_t nCols = 20;
    const size_t nDims = 2; // 2-dimension vector
    size_t buffer[nRows][nCols][nDims];
    size_t *d_buffer;

    // Method of calculating multi-dimensional kernel calls
    dim3 dimBlock(32,32); // This is the maximum as per CUDA 2.x
    dim3 dimGrid( // Method of calculating the number of blocks to use
        (nCols + dimBlock.x - 1) / dimBlock.x,
        (nRows + dimBlock.y - 1) / dimBlock.y,
        nDims);

    memset(buffer, 0, sizeof(buffer));
    hipMalloc(&d_buffer,sizeof(size_t) * nRows * nCols * 2);
    hipMemcpy(d_buffer, buffer, sizeof(buffer), hipMemcpyHostToDevice);
    thread_idx_ND<<<dimGrid,dimBlock>>>(d_buffer, nRows, nCols, nDims);
    hipMemcpy(buffer, d_buffer, sizeof(buffer), hipMemcpyDeviceToHost);
    //hipError_t err = hipDeviceSynchronize();
    //printf("%d\n",err);
    for (size_t idx = 0; idx < nRows; idx++)
    {
        for (size_t jdx = 0; jdx < nCols; jdx++)
        {
            printf("(%d, %d) ", buffer[idx][jdx][0], buffer[idx][jdx][1]);
        }
        printf("\n");
    }
    hipFree(d_buffer);
}

/// @brief 
/// @param dim_x 
/// @param dim_y 
/// @param data 
/// @return 


void test_advect()
{
    size_t idx, jdx;
    const size_t nRows = 5;
    const size_t nCols = 5;

    dim3 dimBlock(32,32); // This is the maximum as per CUDA 2.x
    dim3 dimGrid2( // Method of calculating the number of blocks to use
        (nCols + dimBlock.x - 1) / dimBlock.x,
        (nRows + dimBlock.y - 1) / dimBlock.y);
#if 0
    dim3 dimGrid3( // Method of calculating the number of blocks to use
        (nCols + dimBlock.x - 1) / dimBlock.x,
        (nRows + dimBlock.y - 1) / dimBlock.y,
        2);
#endif
    float pdata[nRows][nCols][2] =
    {
        {{0.0,0.0},{0.0,0.0},{0.0,0.0},{0.0,0.0},{0.0,0.0}},
        {{0.0,0.0},{0.0,0.0},{0.0,0.0},{0.0,0.0},{0.0,0.0}},
        {{0.0,0.0},{0.0,0.0},{-1.0,-1.0},{0.0,0.0},{0.0,0.0}},
        {{0.0,0.0},{0.0,0.0},{0.0,0.0},{0.0,0.0},{0.0,0.0}},
        {{0.0,0.0},{0.0,0.0},{0.0,0.0},{0.0,0.0},{0.0,0.0}}
    };
    float *ddata;
    hipMalloc(&ddata, sizeof(pdata));

    hipMemcpy(ddata, pdata, sizeof(pdata), hipMemcpyHostToDevice);
    kernel_advect<<<dimGrid2, dimBlock>>>(nCols, nRows, ddata);
    hipMemcpy(pdata, ddata, sizeof(pdata), hipMemcpyDeviceToHost);
    for ( idx = 0; idx < nRows; idx++)
    {
        for ( jdx = 0; jdx < nCols; jdx++)
        {
            printf("(%f, %f) ", pdata[idx][jdx][0], pdata[idx][jdx][1]);
        }
        printf("\n");
    }

    unsigned int bgr[nRows][nCols];
    memset(bgr, 0, sizeof(bgr));
    unsigned int *dbgr;
    hipMalloc(&dbgr, sizeof(bgr));

    hipMemcpy(dbgr, bgr, sizeof(bgr), hipMemcpyHostToDevice);
    kernel_gradient<<<dimGrid2, dimBlock>>>(ddata, dbgr, nCols, nRows);
    hipMemcpy(bgr, dbgr, sizeof(bgr), hipMemcpyDeviceToHost);

    for ( idx = 0; idx < nRows; idx++)
    {
        for ( jdx = 0; jdx < nCols; jdx++)
        {
            printf("0x%X ", bgr[idx][jdx]);
        }
        printf("\n");
    }

    hipFree(ddata);
    hipFree(dbgr);
}

int main()
{
    //test_2D_dim();
    //test_ND_dim();
    test_advect();
    return 0;
}