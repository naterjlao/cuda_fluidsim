#include "hip/hip_runtime.h"
#include <stdio.h>
#include "include/fluid_sim.cuh"

__global__ void thread_idx_2D(size_t *buffer, const size_t nCols)
{
    // Example of row, col access into 2D arrays
    const size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    const size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    buffer[row * nCols + col] = col;
}

void test_2D_dim()
{
    const size_t nRows = 512;
    const size_t nCols = 512;
    size_t buffer[nRows][nCols];
    size_t *d_buffer;

    // Method of calculating multi-dimensional kernel calls
    dim3 dimBlock(32, 32); // This is the maximum as per CUDA 2.x
    dim3 dimGrid( // Method of calculating the number of blocks to use
        (nCols + dimBlock.x - 1) / dimBlock.x,
        (nRows + dimBlock.y - 1) / dimBlock.y);

    memset(buffer, 0, sizeof(buffer));
    hipMalloc(&d_buffer,sizeof(size_t) * nRows *nCols);
    hipMemcpy(d_buffer, buffer, sizeof(buffer), hipMemcpyHostToDevice);
    thread_idx_2D<<<dimGrid,dimBlock>>>(d_buffer, nCols);
    hipMemcpy(buffer, d_buffer, sizeof(buffer), hipMemcpyDeviceToHost);
    for (size_t idx = 0; idx < nRows; idx++)
    {
        for (size_t jdx = 0; jdx < nCols; jdx++)
            printf("%d ", buffer[idx][jdx]);
        printf("\n");
    }
    hipFree(d_buffer);
}

#if 0
__global__ void thread_idx_3D(size_t *buffer, const size_t nCols)
{
    // Example of row, col access into 2D arrays
    const size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    buffer[row * nCols + col] = col;
}
#endif

/// @brief 
/// @param dim_x 
/// @param dim_y 
/// @param data 
/// @return 
__global__ void kernel_advect(
    const size_t dim_x,
    const size_t dim_y,
    float* data
)
{
    const size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    const size_t rdx = 1;
    const float timestep = 0.1;
    const float dissipation = 0.999;

    float dx_new, dy_new;

    advect(dim_x,dim_y,x,y,
        rdx,timestep,dissipation,
        data,data,&dx_new, &dy_new);
}

int main()
{
    test_2D_dim();
    return 0;
}