#include "hip/hip_runtime.h"
#include "include/fluid_sim.cuh"

__global__ void kernel_advect(
    const MatrixDim dim,
    const float *input_data,
    float *output_data,
    const float rdx,
    const float timestep,
    const float dissipation)
{
  const size_t x = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t y = blockIdx.y * blockDim.y + threadIdx.y;
  if ((x < dim.x) && (y < dim.y))
  {
    float dx_new = output_data[matrix_index(x, y, dim, 0)];
    float dy_new = output_data[matrix_index(x, y, dim, 1)];

    advect(dim, x, y, rdx, timestep, dissipation,
           input_data, output_data, &dx_new, &dy_new);

    output_data[matrix_index(x, y, dim, 0)] = dx_new;
    output_data[matrix_index(x, y, dim, 1)] = dy_new;
  }
}

/// @brief
/// @param dim_x Dimension x size of the input matrices
/// @param dim_y Dimension y size of the input matrices
/// @param coord_x Position x coordinate
/// @param coord_y Position y coordinate
/// @param timestep Timestep
/// @param u_matrix Input velocity matrix
/// @param d_matrix Matrix to apply advection
/// @return
__host__ __device__ void advect(
    const MatrixDim dim,
    const size_t x,
    const size_t y,
    const float rdx,
    const float timestep,
    const float dissipation,
    const float *u_matrix,
    const float *d_matrix,
    float *dx_new,
    float *dy_new)
{
  // Trace back the trajectory given the current velocity
  const float px = ((float)x) - rdx * timestep * u_matrix[matrix_index(x, y, dim, 0)];
  const float py = ((float)y) - rdx * timestep * u_matrix[matrix_index(x, y, dim, 1)];

  // Given the traceback position, perform bilinear interpolation
  // using the 4 neighboring points and load into the output result
  bilinear_interpolation(px, py, d_matrix, dim, dx_new, dy_new);
}

//-----------------------------------------------------------------------------
/// @brief
/// @param dim Dimension Specification for Vector Field.
/// @param velocity Input VECTOR Velocity Field
/// @param div Output SCALAR Divergence Field
/// @param halfrdx Divergence Factor Constant
/// @return None.
//-----------------------------------------------------------------------------
__global__ void kernel_divergence(
    const MatrixDim dim,
    const float *velocity,
    float *div,
    const float halfrdx)
{
  const size_t x = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t y = blockIdx.y * blockDim.y + threadIdx.y;
  if ((x < dim.x) && (y < dim.y))
  {
    div[y * dim.x + x] = divergence(x, y, velocity, dim, halfrdx);
  }
}

__host__ __device__ float divergence(
    const size_t x, const size_t y,
    const float *data,
    const MatrixDim dim,
    const float halfrdx)
{
  Vector vN, vS, vE, vW;
  neighbors_vector(x, y, data, dim, &vN, &vS, &vE, &vW);

  return halfrdx * (vE.x - vW.x + vS.y - vN.y);
}

__global__ void kernel_jacobi(
    const MatrixDim dim,
    float *X,
    const float *B,
    const float alpha,
    const float beta)
{
  const size_t x = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t y = blockIdx.y * blockDim.y + threadIdx.y;
  if ((x < dim.x) && (y < dim.y))
  {
    X[y * dim.x + x] = jacobi(x, y, X, B, dim, alpha, beta);
  }
}

__host__ __device__ float jacobi(
    const size_t x, const size_t y,
    const float *X,
    const float *B,
    const MatrixDim dim,
    const float alpha,
    const float beta)
{
  float sN, sS, sE, sW;
  neighbors_scalar(x, y, X, dim, &sN, &sS, &sE, &sW);
  const float sB = B[y * dim.x + x];

  return (sN + sS + sE + sW + alpha * sB) * beta;
}

__global__ void kernel_sboundary(
    const MatrixDim dim,
    float *M,
    const float scale)
{
  const size_t x = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t y = blockIdx.y * blockDim.y + threadIdx.y;

  /// @todo There is probably a more elegant way to do this
  if ((x < dim.x) && (y < dim.y))
  {
    if ((x == 0) && (y == 0)) // North-West Corner
    {
      M[y * dim.x + x] = scale * M[(y + 1) * dim.x + (x + 1)];
    }
    else if ((x == (dim.x - 1)) && (y == 0)) // North-East Corner
    {
      M[y * dim.x + x] = scale * M[(y + 1) * dim.x + (x - 1)];
    }
    else if ((x == 0) && (y == (dim.y - 1))) // South-West Corner
    {
      M[y * dim.x + x] = scale * M[(y - 1) * dim.x + (x + 1)];
    }
    else if ((x == (dim.x - 1)) && (y == (dim.y - 1))) // South-East Corner
    {
      M[y * dim.x + x] = scale * M[(y - 1) * dim.x + (x - 1)];
    }
    else if (x == 0) // West Border
    {
      M[y * dim.x + x] = scale * M[y * dim.x + (x + 1)];
    }
    else if (y == 0) // North Border
    {
      M[y * dim.x + x] = scale * M[(y + 1) * dim.x + x];
    }
    else if (x == (dim.x - 1)) // East Border
    {
      M[y * dim.x + x] = scale * M[y * dim.x + (x - 1)];
    }
    else if (y == (dim.y - 1)) // South Border
    {
      M[y * dim.x + x] = scale * M[(y - 1) * dim.x + x];
    }
  }
}

__global__ void kernel_vboundary(
    const MatrixDim dim,
    float *M,
    const float scale)
{
  const size_t x = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t y = blockIdx.y * blockDim.y + threadIdx.y;

  /// @todo There is probably a more elegant way to do this
  if ((x < dim.x) && (y < dim.y))
  {
    if ((x == 0) && (y == 0)) // North-West Corner
    {
      M[matrix_index(x, y, dim, 0)] = scale * M[matrix_index(x + 1, y + 1, dim, 0)];
      M[matrix_index(x, y, dim, 1)] = scale * M[matrix_index(x + 1, y + 1, dim, 1)];
    }
    else if ((x == (dim.x - 1)) && (y == 0)) // North-East Corner
    {
      M[matrix_index(x, y, dim, 0)] = scale * M[matrix_index(x - 1, y + 1, dim, 0)];
      M[matrix_index(x, y, dim, 1)] = scale * M[matrix_index(x - 1, y + 1, dim, 1)];
    }
    else if ((x == 0) && (y == (dim.y - 1))) // South-West Corner
    {
      M[matrix_index(x, y, dim, 0)] = scale * M[matrix_index(x + 1, y - 1, dim, 0)];
      M[matrix_index(x, y, dim, 1)] = scale * M[matrix_index(x + 1, y - 1, dim, 1)];
    }
    else if ((x == (dim.x - 1)) && (y == (dim.y - 1))) // South-East Corner
    {
      M[matrix_index(x, y, dim, 0)] = scale * M[matrix_index(x - 1, y - 1, dim, 0)];
      M[matrix_index(x, y, dim, 1)] = scale * M[matrix_index(x - 1, y - 1, dim, 1)];
    }
    else if (x == 0) // West Border
    {
      M[matrix_index(x, y, dim, 0)] = scale * M[matrix_index(x + 1, y, dim, 0)];
    }
    else if (y == 0) // North Border
    {
      M[matrix_index(x, y, dim, 1)] = scale * M[matrix_index(x, y + 1, dim, 0)];
    }
    else if (x == (dim.x - 1)) // East Border
    {
      M[matrix_index(x, y, dim, 0)] = scale * M[matrix_index(x - 1, y, dim, 0)];
    }
    else if (y == (dim.y - 1)) // South Border
    {
      M[matrix_index(x, y, dim, 1)] = scale * M[matrix_index(x, y - 1, dim, 0)];
    }
  }
}
